#include "thrust.cuh"

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/sort.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

#include <thrust/sequence.h>
#include <thrust/for_each.h>

#include "mod_range.cuh"

namespace my_thrust {
  // void my_thrust::stable_sort() {
  //     thrust::device_ptr<float> d_ptr = thrust::device_malloc<float>(3);

  //     thrust::device_ptr<float> first = d_ptr;
  //     thrust::device_ptr<float> last  = d_ptr + 3;

  //     d_ptr[0] = 3.0; d_ptr[1] = 2.0; d_ptr[2] = 1.0;
  //     thrust::stable_sort(first, last);

  //     std::cout << d_ptr[0] << ", " << d_ptr[1] << ", " << d_ptr[2] << std::endl;

  //     thrust::device_free(d_ptr);
  // }


  static const int NSORTS = 16000;
  static const int DSIZE = 1000;

  struct Mod {
    int d_;
    int p_;
    Mod(int d) : d_(d) {}
    int operator()() {
      return p_++ / d_;
    }
  };

  thrust::device_vector<int> gen_rand() {
    thrust::host_vector<int> h_data(DSIZE*NSORTS);
    thrust::generate(h_data.begin(), h_data.end(), rand);
    thrust::device_vector<int> d_data = h_data;
    return d_data;
  }

  bool validate(const thrust::device_vector<int> &d1, const thrust::device_vector<int> &d2){
    return thrust::equal(d1.cbegin(), d1.cend(), d2.cbegin());
  }

  void print(const thrust::device_vector<int>& result) {
    std::cout << result[0] << ", " << result[1] << ", " << result[2] << " ... ";
    std::cout << result[DSIZE-3] << ", " << result[DSIZE-2] << ", " << result[DSIZE-1] << std::endl;
    int c = (NSORTS - 1) * DSIZE;
    std::cout << result[c+0] << ", " << result[c+1] << ", " << result[c+2] << " ... ";
    std::cout << result[c+DSIZE-3] << ", " << result[c+DSIZE-2] << ", " << result[c+DSIZE-1] << std::endl;
  }

  template <class Proc>
  void benchmark(Proc proc) {
    thrust::device_vector<int> d_vec = gen_rand();
    auto expect = d_vec;

    for (int i = 0; i < NSORTS; i++) {
      thrust::sort(expect.begin() + (i*DSIZE), expect.begin() + ((i+1)*DSIZE));
    }

    // https://ivanlife.wordpress.com/2011/05/09/time-cuda/
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    proc(d_vec);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);

    print(d_vec);
    printf ("Time for the kernel: %f ms\n", time);
    if (validate(d_vec, expect)) {
      printf("OK!\n");
    } else {
      printf("failed!\n");
    }
  }

  void stable_sort_batch_vector() {
    // ??? count_iterator ???
    benchmark([=](thrust::device_vector<int>& d_vec) {
        // thrust::host_vector<int> h_segments(DSIZE*NSORTS);
        // thrust::generate(h_segments.begin(), h_segments.end(), Mod(DSIZE));
        // thrust::device_vector<int> d_segments = h_segments;
        typedef thrust::device_vector<int>::iterator Iterator;
        mod_range<Iterator> d_segments(d_vec.begin(), d_vec.end(), DSIZE);

        thrust::stable_sort_by_key(d_vec.begin(), d_vec.end(), d_segments.begin());
        // thrust::stable_sort_by_key(d_segments.begin(), d_segments.end(), d_vec.begin());
      });
  }



  struct SortFunctor
  {
    thrust::device_ptr<int> data;
    int dsize;
    __host__ __device__
    void operator()(int start_idx)
    {
      thrust::sort(thrust::device, data+(dsize*start_idx), data+(dsize*(start_idx+1)));
    }
  };

  void stable_sort_batch_nested() {
    benchmark([=](thrust::device_vector<int>& d_vec) {
        hipDeviceSetLimit(hipLimitMallocHeapSize, (16*DSIZE*NSORTS));
        thrust::device_vector<int> d_result3 = gen_rand();
        SortFunctor f = {d_result3.data(), DSIZE};
        thrust::device_vector<int> idxs(NSORTS);
        thrust::sequence(idxs.begin(), idxs.end());
        thrust::for_each(idxs.begin(), idxs.end(), f);
      });
  }

} // namespace
